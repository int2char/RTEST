#include "hip/hip_runtime.h"
#include "GraphPath.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include<vector>
#include<algorithm>
#include <utility>
#include <time.h>
#include<math.h>
#include"service.h"
#include"dijkstra.h"
#include"BFS.h"
#include"taskPath.h"
#include"const.h"
#include"routemask.h"
#include"PathArrange.h"
#include<fstream>
using namespace std;
#define threadsize 256

bool UDgreater(pair<int, float> elem1, pair<int, float> elem2)
{
	return elem1.second > elem2.second;
}
bool UPGservice(service s1, service s2)
{
	return s1.d>s2.d;
}
bool cmp(float a, float b)
{
	return a<b;
}
__global__ void bellmanHigh(Edge *edge, int *m, float *c, int*p, float*lambda, int*mask, int stillS)
{
	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= stillS)return;
	i = mask[i];
	int head = edge[tid].head;
	int tail = edge[tid].tail;
	int biao = head*Task + i;
	float val = c[tail*Task + i]+1 +lambda[tid];
	if (c[biao] >val){
		*m = 1;
		c[biao] = val;
	}
}
__global__ void color(Edge *edge, int *m, float *c, int*p, float*lambda, int *mask, int stillS){

	int tid = blockIdx.y;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >= stillS)return;
	i = mask[i];
	int head = edge[tid].head;
	int tail = edge[tid].tail;
	int biao = head*Task + i;
	float val = c[tail*Task + i]+1+lambda[tid];// * pd[i];
	if (c[biao] == val){
		p[biao] = tid;
	}
}
__global__ void ChangePameterC(int*p, float*d, int* st, int taskSize, int n){
	int tid = blockIdx.y;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i >= taskSize || tid >= n)return;
	int biao = tid*taskSize + i;
	d[biao] = (st[i] == tid) ? 0.0 : 10000000000.0;
	p[biao] = -1;


}
void GraphPath::Copy2GPU(std::vector<service> &s){
	for (int i = 0; i < Task; i++)
	{
		st[i] = s[i].s;
		te[i] = s[i].t;
		pd[i] = (float)s[i].d;
	}
	for (int i = 0; i < Task; i++)
		mask[i] = i;
	for (int i = 0; i < EDge; i++)
		lambda[i] = 0;
	hipMemcpy(dev_st, st, Task*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_te, te, Task*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_lambda, lambda, EDge*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mask, mask, Task*sizeof(int), hipMemcpyHostToDevice);
}


GraphPath::GraphPath(Graph&_G):G(_G),StoreRoute(Task, vector<int>(1,-1)), BestRoute(Task, vector<int>())
{
	hipMalloc(&dev_edge, sizeof(Edge)*EDge);
	hipMemcpy(dev_edge, G.incL, EDge* sizeof(Edge), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_st, Task*sizeof(int));
	hipMalloc((void**)&dev_te, Task*sizeof(int));
	hipMalloc((void**)&dev_pd, Task*sizeof(float));
	hipMalloc((void**)&dev_lambda, EDge*sizeof(float));
	hipMalloc((void**)&dev_mask, Task*sizeof(int));
	hipMalloc((void**)&dev_d, Task*NODE* sizeof(float));
	hipMalloc((void**)&dev_p, Task*NODE* sizeof(int));
	hipMalloc(&dev_m, sizeof(int));
	st = new int[Task*sizeof(int)];
	te = new int[Task*sizeof(int)];
	pd = new float[Task*sizeof(float)];
	d = (float*)malloc(Task*NODE*sizeof(float));
	pre = (int*)malloc(Task*NODE*sizeof(int));
	lambda = new float[EDge*sizeof(float)];
	mask = new int[Task];
	mark = new int(1);
	capacity = (float*)malloc(EDge*sizeof(float));
	for (int i = 0; i < NODE; i++)
		{
			for (int j = 0; j < Task; j++)
			{
				if (st[j] == i)
				{
					d[i*Task + j] = 0.0;
					pre[i*Task + j] = -1;
				}
				else
				{
					d[i*Task + j] = 100000.0;
					pre[i*Task + j] = -1;
				}
			}
		}
	hipMemcpy(dev_d, d, Task*NODE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, pre, Task*NODE*sizeof(int), hipMemcpyHostToDevice);
}
vector<pair<string,float> > GraphPath::bellmanFordCuda(vector<service>&ser,ostream& Out) {
	printf("Lagrange parrel searching..............\n");
	srand(time(NULL));
	vector<pair<string,float>> rdata;
	return rdata;
}
void GraphPath::CudaFree(){
	hipFree(dev_st);
	hipFree(dev_te);
	hipFree(dev_pd);
	hipFree(dev_lambda);
	hipFree(dev_mask);
	hipFree(dev_d);
	hipFree(dev_p);
	hipFree(dev_m);

}
GraphPath::~GraphPath()
{
	CudaFree();
	delete[] st;
	delete[] te;
	delete[] pd;
	free(d);
	free(pre);
	delete[] lambda;
	delete[] mask;
	delete mark;
	free(capacity);
}



